#include "hip/hip_runtime.h"
/**
 * Created by desmond <desmond.yao@buaa.edu.cn> on 2018-11-18
 */
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this project

//Each channel Red, Blue, Green, and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color. This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

#ifndef GRAYSCALE_H__
#define GRAYSCALE_H__

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "./Utils/utils.h"

#define  descale(x,n)  (((x) + (1 << ((n)-1))) >> (n))

#define  SCALE  14
#define  cR  (int)(0.299*(1 << SCALE) + 0.5)
#define  cG  (int)(0.587*(1 << SCALE) + 0.5)
#define  cB  ((1 << SCALE) - cR - cG)

// CUDA kernel which is run in parallel by many GPU threads.
__global__
void rgbaToGreyscaleCudaKernel(const uchar4* const rgbaImage,
		unsigned char* const greyImage,
		const int numRows, const int numCols)
{
	//First create a mapping from the 2D block and grid locations
	//to an absolute 2D location in the image, then use that to
	//calculate a 1D offset
	const long pointIndex = threadIdx.x + blockDim.x*blockIdx.x;

	if(pointIndex<numRows*numCols) { // this is necessary only if too many threads are started
		uchar4 const imagePoint = rgbaImage[pointIndex];
		greyImage[pointIndex] = descale(cR*imagePoint.x + cG*imagePoint.y  + cB*imagePoint.z, SCALE);
	}
}

// Parallel implementation for running on GPU using multiple threads.
void rgbaToGreyscaleCuda(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
		unsigned char* const d_greyImage, const size_t numRows, const size_t numCols)
{
	const int blockThreadSize = 1024;
	const int numberOfBlocks = 1 + ((numRows*numCols - 1) / blockThreadSize); // a/b rounded up
    printf("numberOfBlocks %d\n", numberOfBlocks);
	const dim3 blockSize(blockThreadSize, 1, 1);
	const dim3 gridSize(numberOfBlocks , 1, 1);
	rgbaToGreyscaleCudaKernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);


	//----------------------------------------calculate optimized grid and block size-------------------------------------------------------
    int minGridSize = 0;
    int blockSize_ = 0;      // The launch configurator returned block size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_, (void* )rgbaToGreyscaleCudaKernel, 0, numRows * numCols);

    int gridSize_ = (numRows * numCols + blockSize_ - 1) / blockSize_;

    printf("Launch grids of size %d. Launch blocks of size %d\n", gridSize_, blockSize_);

}

// Serial implementation for running on CPU using a single thread.
void rgbaToGreyscaleCpu(const uchar4* const rgbaImage, unsigned char *const greyImage,
		const size_t numRows, const size_t numCols)
{
	for (size_t r = 0; r < numRows; ++r) {
		for (size_t c = 0; c < numCols; ++c) {
			const uchar4 rgba = rgbaImage[r * numCols + c];
			const float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
			greyImage[r * numCols + c] = channelSum;
		}
	}
}

#endif  /* GRAYSCALE_H__ */
